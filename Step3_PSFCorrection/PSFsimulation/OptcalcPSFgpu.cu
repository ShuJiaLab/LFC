#include "hip/hip_runtime.h"
/**********************************************
 * This is the source code for PSF calculation *
 * using AMD/NVDA GPU based on OpenCL frame.   *
 * 											   *
 * Maintained by Xuanwen Hua				   *
 * (xwghua@gmail.com)						   *
***********************************************/


#include <math.h>


/* Identify the sub-functions that will be used in the following codes.
/////////////////////////////////////////////////////////
// Here is the kernel function denoted as "__kernel"
// which means it can only run on the OpenCL platform.
**********************************************************/

__global__ void calcPSFgpu(double *pdev_p1,
                        double *pdev_p2,
                        double *pdev_p3,
                        double *pdev_fobj,
                        double *pdev_k,
                        double *pdev_alpha, //*1e9
                        double *pdev_M,
                        double *pdev_wavelen,
                        int *pdev_boundary,
                        int *pdev_centerPT,
                        double *pn_obj,
                        double *pn_sample,
                        //__global float* dev_zeroline,
                        double *dev_xspace, //*1e6
                        double *dev_yspace, //*1e6
                        double *dev_gkx,
                        double *dev_gkw1,
                        double *dev_pattern_quart_re,
                        double *dev_pattern_quart_im) {
/* 
Note that the pattern_quart_xx is from the boundary(or boundary-1 in C
indeces) to the centerPT (or the centerPT-1 in C indeces), and calculation
only works when the row & col indeces meet the 1/8 limitation.
This scheme actually minimize the calculation consumption in case of large
matrices being introduced.
*/
    // double dev_p1 = *pdev_p1;
    // double dev_p2 = *pdev_p2;
    double dev_p3 = *pdev_p3;
    // double dev_fobj = *pdev_fobj;
    double dev_k = *pdev_k;
    double dev_alpha = *pdev_alpha;
    double dev_M = *pdev_M;
    // double dev_wavelen = *pdev_wavelen;
    int dev_boundary = *pdev_boundary;
    int dev_centerPT = *pdev_centerPT;
    double n_obj = *pn_obj;
    double n_sample = *pn_sample;

    __device__ double cos(double x);
    __device__ double sin(double x);
    __device__ double pow(double x, double y);
    __device__ double sqrt(double x);
    __device__ __CUDA_MATH_CRTIMP double j0(double x);
    __device__ __CUDA_MATH_CRTIMP double j1(double x);
    __device__ __CUDA_MATH_CRTIMP double jn(int bessel_n, double x);
    //printf("########## Kernel calculation started ##########");

    int col_pt_total = dev_centerPT - dev_boundary;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    long int gid = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x) + threadIdx.x;

    
    
    int rowpt = gid/col_pt_total; // row a
    int colpt = gid%col_pt_total; // col b
    
    double dev_gkx2i,tau_si,tau_pi,kphid;
    double rim_d = 1e-7; //100 nm
    // double rim_d = -10e-6; //10 um
    // double rim_d = 30e-6; //30 um
    // double rim_d = 50e-6; //50 um
    // double rim_d = 70e-6; //70 um
    // double rim_d = 90e-6; //90 um
    // double rim_d = 110e-6; //110 um
    // double rim_d = 130e-6; //130 um
    // double rim_d = 150e-6; //150 um
    // double rim_d = 170e-6; //170 um
    // printf("col_pt_total = %d\n", col_pt_total);
    // printf("rowpt, colpt, gid = %d, %d, %d\n", rowpt, colpt, gid);
    if ((rowpt<dev_centerPT) && (colpt<dev_centerPT))
    {
        // printf("%d | %d,%d | %lf,%lf,\n",gid,
        //     rowpt, colpt,
        //     dev_xspace[rowpt],dev_yspace[colpt]);

        dev_pattern_quart_re[gid] = 0.0;
        dev_pattern_quart_im[gid] = 0.0;

        if (colpt>=rowpt)
        { // identify the 1/8 boundary
            double x1 = dev_xspace[rowpt + dev_boundary ]; //*1e6
            double x2 = dev_yspace[colpt + dev_boundary ]; //*1e6
            //printf("%lf,%lf\n",x1,x2);
            double xL2normsq = (sqrt(x1*x1 + x2*x2))/(dev_M*1e6);
            //printf("%lf\n", sqrt(x1*x1 + x2*x2));
            double v = dev_k*xL2normsq*sin(dev_alpha);
            double u = 4*dev_k*1e-6 / n_obj * n_sample *dev_p3*pow(sin(dev_alpha/2),2);
            // double phi_p = atan(x2/x1);
            //printf("%lf, %lf\n", u,v);
            //printf("===== ");

            double U0_re = 0; int i;
            for (i=0; i<201; i++){
                // printf("i = %d\n", i);
                dev_gkx2i = asin(sin(dev_gkx[i]) * n_obj / n_sample);
                tau_si = 2 * sin(dev_gkx2i) * cos(dev_gkx[i]) / sin(dev_gkx[i] + dev_gkx2i);
                tau_pi = tau_si / cos (dev_gkx[i] - dev_gkx2i);
                kphid = (0.0-dev_k) /n_obj * rim_d * (n_obj * cos(dev_gkx[i]) - n_sample * cos(dev_gkx2i));
                // printf("dev_gkx2i,tau_si,tau_pi,kphid = (%f,%f, %f,%f)\n",dev_gkx2i,tau_si,tau_pi,kphid);
                //////////////////////////////////////////////
                U0_re = U0_re + (sqrt(cos(dev_gkx[i])))*
                (tau_si+tau_pi * cos(dev_gkx2i))*
                (cos(kphid + (u/2)*(pow(sin(dev_gkx2i/2),2))/pow(sin(dev_alpha/2),2)))*
                (j0(sin(dev_gkx[i])/sin(dev_alpha)*v))*(sin(dev_gkx[i])) *
                dev_gkw1[i];
                // if (isnan(U0_re)==1){
                //     printf("i, dev_gkx[i],dev_gkx2i,tau_si,tau_pi,kphid = (%d,%f,%f,%f, %f,%f)\n",i,dev_gkx[i],dev_gkx2i,tau_si,tau_pi,kphid);
                // }
                // else
                //     ;
            }
            

            double U0_im = 0; int j;
            for (j=0; j<201; j++){
                // printf("j = %d\n", j);
                dev_gkx2i = asin(sin(dev_gkx[j]) * n_obj / n_sample);
                tau_si = 2 * sin(dev_gkx2i) * cos(dev_gkx[j]) / sin(dev_gkx[j] + dev_gkx2i);
                tau_pi = tau_si / cos (dev_gkx[j] - dev_gkx2i);
                kphid = (0.0-dev_k) /n_obj * rim_d * (n_obj * cos(dev_gkx[j]) - n_sample * cos(dev_gkx2i));
                //////////////////////////////////////////////
                U0_im = U0_im + (sqrt(cos(dev_gkx[j])))*
                (tau_si+tau_pi * cos(dev_gkx2i))*
                (sin(kphid + (u/2)*(pow(sin(dev_gkx2i/2),2))/pow(sin(dev_alpha/2),2)))*
                (j0(sin(dev_gkx[j])/sin(dev_alpha)*v))*(sin(dev_gkx[j])) *
                dev_gkw1[j];
            }

            // double U0_re2 = 0; int ii;
            // for (ii=0; ii<201; ii++){
            //     // printf("i = %d\n", i);
            //     dev_gkx2i = asin(sin(dev_gkx[ii]) * n_obj / n_sample);
            //     tau_si = 2 * sin(dev_gkx2i) * cos(dev_gkx[ii]) / sin(dev_gkx[ii] + dev_gkx2i);
            //     tau_pi = tau_si / cos (dev_gkx[ii] - dev_gkx2i);
            //     kphid = -dev_k /n_obj * rim_d * (n_obj * cos(dev_gkx[ii]) - n_sample * cos(dev_gkx2i));
            //     //////////////////////////////////////////////
            //     U0_re2 = U0_re2 + (sqrt(cos(dev_gkx[ii])))*
            //     (tau_pi * sin(dev_gkx2i))*
            //     (cos(kphid + (u/2)*(pow(sin(dev_gkx2i/2),2))/pow(sin(dev_alpha/2),2)))*
            //     (j0(sin(dev_gkx[ii])/sin(dev_alpha)*v))*(sin(dev_gkx[ii])) *
            //     dev_gkw1[ii];
            // //     //printf("U0_re = %lf\n", U0_re);
            // }

            // double U0_im2 = 0; int jj;
            // for (jj=0; jj<201; jj++){
            //     // printf("j = %d\n", j);
            //     dev_gkx2i = asin(sin(dev_gkx[jj]) * n_obj / n_sample);
            //     tau_si = 2 * sin(dev_gkx2i) * cos(dev_gkx[jj]) / sin(dev_gkx[jj] + dev_gkx2i);
            //     tau_pi = tau_si / cos (dev_gkx[jj] - dev_gkx2i);
            //     kphid = -dev_k /n_obj * rim_d * (n_obj * cos(dev_gkx[jj]) - n_sample * cos(dev_gkx2i));
            //     //////////////////////////////////////////////
            //     U0_im2 = U0_im2 + (sqrt(cos(dev_gkx[jj])))*
            //     (tau_pi * sin(dev_gkx2i))*
            //     (sin(kphid + (u/2)*(pow(sin(dev_gkx2i/2),2))/pow(sin(dev_alpha/2),2)))*
            //     (j0(sin(dev_gkx[jj])/sin(dev_alpha)*v))*(sin(dev_gkx[jj])) *
            //     dev_gkw1[jj];
            // }

            double U0_re3 = 0; int iii;
            for (iii=0; iii<201; iii++){
                // printf("i = %d\n", i);
                dev_gkx2i = asin(sin(dev_gkx[iii]) * n_obj / n_sample);
                tau_si = 2 * sin(dev_gkx2i) * cos(dev_gkx[iii]) / sin(dev_gkx[iii] + dev_gkx2i);
                tau_pi = tau_si / cos (dev_gkx[iii] - dev_gkx2i);
                kphid = (0.0-dev_k) /n_obj * rim_d * (n_obj * cos(dev_gkx[iii]) - n_sample * cos(dev_gkx2i));
                //////////////////////////////////////////////
                U0_re3 = U0_re3 + (sqrt(cos(dev_gkx[iii])))*
                (tau_si-tau_pi * cos(dev_gkx2i))*
                (cos(kphid + (u/2)*(pow(sin(dev_gkx2i/2),2))/pow(sin(dev_alpha/2),2)))*
                (jn(2,sin(dev_gkx[iii])/sin(dev_alpha)*v))*(sin(dev_gkx[iii])) *
                // (j1(sin(dev_gkx[iii])/sin(dev_alpha)*v))*(sin(dev_gkx[iii])) *
                dev_gkw1[iii];
            //     //printf("U0_re = %lf\n", U0_re);
            }

            double U0_im3 = 0; int jjj;
            for (jjj=0; jjj<201; jjj++){
                // printf("j = %d\n", j);
                dev_gkx2i = asin(sin(dev_gkx[jjj]) * n_obj / n_sample);
                tau_si = 2 * sin(dev_gkx2i) * cos(dev_gkx[jjj]) / sin(dev_gkx[jjj] + dev_gkx2i);
                tau_pi = tau_si / cos (dev_gkx[jjj] - dev_gkx2i);
                kphid = (0.0-dev_k) /n_obj * rim_d * (n_obj * cos(dev_gkx[jjj]) - n_sample * cos(dev_gkx2i));
                //////////////////////////////////////////////
                U0_im3 = U0_im3 + (sqrt(cos(dev_gkx[jjj])))*
                (tau_si-tau_pi * cos(dev_gkx2i))*
                (sin(kphid + (u/2)*(pow(sin(dev_gkx2i/2),2))/pow(sin(dev_alpha/2),2)))*
                (jn(2,sin(dev_gkx[jjj])/sin(dev_alpha)*v))*(sin(dev_gkx[jjj])) *
                dev_gkw1[jjj];
            }
            /*************************************/
            // double U0_re = gk_re(dev_gkx, dev_gkw1, dev_alpha, u, v);
            // double U0_im = gk_im(dev_gkx, dev_gkw1, dev_alpha, u, v);

            /***********************************/
            dev_pattern_quart_re[gid] = U0_re-U0_re3; //*1e18
            dev_pattern_quart_im[gid] = U0_im-U0_im3; //*1e18
            //printf("Koi_re, Koi_im = (%f, %f)\n",Koi_re,Koi_im);
            //printf("dev_pattern_quart_re, dev_pattern_quart_im = (%f, %f)\n",U0_re*Koi_re - U0_im*Koi_im, U0_re*Koi_im + U0_im*Koi_re);
            //printf("u,v: %lf, %lf\n",u,v);
            // printf("U0_re,U0_re3, U0_im,U0_im3 = (%f,%f, %f,%f)\n",U0_re,U0_re3,U0_im,U0_im3);
            
            
        }
        else
            ;
    }
    
    
    //printf("########## kernel completed! ##########\n");
    
}



/************************
// ------------------------------
// This part is for reference as it's a CPU scheme
// ------------------------------
    for (unsigned int a = dev_boundary,a<=dev_centerPT,a++){
        int x1 = dev_xspace[a-1];
        float* patternLine = dev_zeroline;
        for (unsigned int b = a,b<=dev_centerPT,b++){
            int x2 = dev_yspace[b-1];
            float xL2normsq = (sqrt(pow(x1+dev_M*dev_p1,2) + pow(x2+dev_M*dev_p2,2)))/dev_M;
            float v = dev_k*xL2normsq*sin(dev_alpha);
            float u = 4*dev_k*dev_p3*pow(sin(dev_alpha/2),2);
            float Koi_re = dev_M/(pow(dev_fobj*dev_wavelen,2))*cos(u/(4*pow(sin(dev_alpha/2),2)));
            float Koi_im = dev_M/(pow(dev_fobj*dev_wavelen,2))*cos(u/(4*pow(sin(dev_alpha/2),2)));
            float U_re = NumInt_re(dev_alpha,u,v,0,dev_alpha,1e-4);
            float U_im = NumInt_im(dev_alpha,u,v,0,dev_alpha,1e-4);
            float U0_re = U_re*Koi_re - U_im*Koi_im;
            float U0_im = U_re*Koi_im + U_im*Koi_re;

        }
    }
***********************/