#include "hip/hip_runtime.h"
/**********************************************
 * This is the source code for PSF calculation *
 * using AMD/NVDA GPU based on OpenCL frame.   *
 * 											   *
 * Maintained by Xuanwen Hua				   *
 * (xwghua@gmail.com)						   *
***********************************************/


#include <math.h>


/* Identify the sub-functions that will be used in the following codes.
/////////////////////////////////////////////////////////
// Here is the kernel function denoted as "__kernel"
// which means it can only run on the OpenCL platform.
**********************************************************/

__global__ void calcPSFgpu(double *pdev_p1,
                        double *pdev_p2,
                        double *pdev_p3,
                        double *pdev_fobj,
                        double *pdev_k,
                        double *pdev_alpha, //*1e9
                        double *pdev_M,
                        double *pdev_wavelen,
                        int *pdev_boundary,
                        int *pdev_centerPT,
                        //__global float* dev_zeroline,
                        double *dev_xspace, //*1e6
                        double *dev_yspace, //*1e6
                        double *dev_gkx,
                        double *dev_gkw1,
                        double *dev_pattern_quart_re,
                        double *dev_pattern_quart_im) {
/* 
Note that the pattern_quart_xx is from the boundary(or boundary-1 in C
indeces) to the centerPT (or the centerPT-1 in C indeces), and calculation
only works when the row & col indeces meet the 1/8 limitation.
This scheme actually minimize the calculation consumption in case of large
matrices being introduced.
*/
    // double dev_p1 = *pdev_p1;
    // double dev_p2 = *pdev_p2;
    double dev_p3 = *pdev_p3;
    // double dev_fobj = *pdev_fobj;
    double dev_k = *pdev_k;
    double dev_alpha = *pdev_alpha;
    double dev_M = *pdev_M;
    // double dev_wavelen = *pdev_wavelen;
    int dev_boundary = *pdev_boundary;
    int dev_centerPT = *pdev_centerPT;

    __device__ double cos(double x);
    __device__ double sin(double x);
    __device__ double pow(double x, double y);
    __device__ double sqrt(double x);
    __device__ __CUDA_MATH_CRTIMP double j0(double x);
    //printf("########## Kernel calculation started ##########");

    int col_pt_total = dev_centerPT - dev_boundary;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    long int gid = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x) + threadIdx.x;

    
    
    int rowpt = gid/col_pt_total; // row a
    int colpt = gid%col_pt_total; // col b
    
    // printf("col_pt_total = %d\n", col_pt_total);
    // printf("rowpt, colpt, gid = %d, %d, %d\n", rowpt, colpt, gid);
    if ((rowpt<dev_centerPT) && (colpt<dev_centerPT))
    {
        // printf("%d | %d,%d | %lf,%lf,\n",gid,
        //     rowpt, colpt,
        //     dev_xspace[rowpt],dev_yspace[colpt]);

        dev_pattern_quart_re[gid] = 0.0;
        dev_pattern_quart_im[gid] = 0.0;

        if (colpt>=rowpt)
        { // identify the 1/8 boundary
            double x1 = dev_xspace[rowpt + dev_boundary ]; //*1e6
            double x2 = dev_yspace[colpt + dev_boundary ]; //*1e6
            //printf("%lf,%lf\n",x1,x2);
            double xL2normsq = (sqrt(x1*x1 + x2*x2))/(dev_M*1e6);
            //printf("%lf\n", sqrt(x1*x1 + x2*x2));
            double v = dev_k*xL2normsq*sin(dev_alpha);
            double u = 4*dev_k*1e-6*dev_p3*pow(sin(dev_alpha/2),2);
            //printf("%lf, %lf\n", u,v);
            //printf("===== ");

            double U0_re = 0; int i;
            for (i=0; i<201; i++){
                // printf("i = %d\n", i);
                U0_re = U0_re + (sqrt(cos(dev_gkx[i])))*(1+cos(dev_gkx[i]))*
                (cos((u/2)*(pow(sin(dev_gkx[i]/2),2))/pow(sin(dev_alpha/2),2)))*
                (j0(sin(dev_gkx[i])/sin(dev_alpha)*v))*(sin(dev_gkx[i])) *
                dev_gkw1[i];
            //     //printf("U0_re = %lf\n", U0_re);
            }

            double U0_im = 0; int j;
            for (j=0; j<201; j++){
                // printf("j = %d\n", j);
                U0_im = U0_im + (sqrt(cos(dev_gkx[j])))*(1+cos(dev_gkx[j]))*
                (sin((u/2)*(pow(sin(dev_gkx[j]/2),2))/pow(sin(dev_alpha/2),2)))*
                (j0(sin(dev_gkx[j])/sin(dev_alpha)*v))*(sin(dev_gkx[j])) *
                dev_gkw1[j];
            }


            /*************************************/
            // double U0_re = gk_re(dev_gkx, dev_gkw1, dev_alpha, u, v);
            // double U0_im = gk_im(dev_gkx, dev_gkw1, dev_alpha, u, v);

            /***********************************/
            dev_pattern_quart_re[gid] = U0_re; //*1e18
            dev_pattern_quart_im[gid] = U0_im; //*1e18

            //printf("Koi_re, Koi_im = (%f, %f)\n",Koi_re,Koi_im);
            //printf("dev_pattern_quart_re, dev_pattern_quart_im = (%f, %f)\n",U0_re*Koi_re - U0_im*Koi_im, U0_re*Koi_im + U0_im*Koi_re);
            //printf("u,v: %lf, %lf\n",u,v);
            //printf("U0_re, U0_im = (%f, %f)\n",U0_re,U0_im);
            
            
        }
        else
            ;
    }
    
    
    //printf("########## kernel completed! ##########\n");
    
}



/************************
// ------------------------------
// This part is for reference as it's a CPU scheme
// ------------------------------
    for (unsigned int a = dev_boundary,a<=dev_centerPT,a++){
        int x1 = dev_xspace[a-1];
        float* patternLine = dev_zeroline;
        for (unsigned int b = a,b<=dev_centerPT,b++){
            int x2 = dev_yspace[b-1];
            float xL2normsq = (sqrt(pow(x1+dev_M*dev_p1,2) + pow(x2+dev_M*dev_p2,2)))/dev_M;
            float v = dev_k*xL2normsq*sin(dev_alpha);
            float u = 4*dev_k*dev_p3*pow(sin(dev_alpha/2),2);
            float Koi_re = dev_M/(pow(dev_fobj*dev_wavelen,2))*cos(u/(4*pow(sin(dev_alpha/2),2)));
            float Koi_im = dev_M/(pow(dev_fobj*dev_wavelen,2))*cos(u/(4*pow(sin(dev_alpha/2),2)));
            float U_re = NumInt_re(dev_alpha,u,v,0,dev_alpha,1e-4);
            float U_im = NumInt_im(dev_alpha,u,v,0,dev_alpha,1e-4);
            float U0_re = U_re*Koi_re - U_im*Koi_im;
            float U0_im = U_re*Koi_im + U_im*Koi_re;

        }
    }
***********************/